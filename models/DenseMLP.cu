//
// Created by DanielSun on 11/22/2021.
//

#include "DenseMLP.cuh"
#include "../io/Reader.cuh"
#include "../utils/logger.cuh"

int readDataset(const string& path0, vector<Matrix::Matrix2d*>* data, vector<Matrix::Matrix2d*>* label,
                 DenseMLP::Config cfg, int labelIndex, int count) {
    intptr_t hFile = 0;
    struct _finddata_t fileInfo{};
    string p;
    string* paths;
    vector<string> temp;
    vector<Matrix::Matrix2d *> buf;
    unsigned char* buffer;
    unsigned char* bufCuda;

    if ((hFile = _findfirst(p.assign(path0).append("\\*").c_str(), &fileInfo)) != -1){
        while (_findnext(hFile, &fileInfo) == 0){
            temp.push_back(p.assign(path0).append("\\").append(fileInfo.name));
        }
    }
    hipHostMalloc((void**)&paths, sizeof(string)*temp.size());
    hipHostMalloc((void**)&buffer, sizeof(char)*cfg.CPU_THREADS*cfg.BMP_READ_SIZE);
    hipMalloc((void**)&bufCuda, sizeof(char)*cfg.CPU_THREADS*cfg.BMP_READ_SIZE);
    for(int i=1; i< temp.size(); ++i){
        paths[i-1] = temp[i];
    }

    for (int i=0;i< temp.size();i++) {
        Matrix::Matrix2d* matData;
        hipHostMalloc(&matData,sizeof(Matrix::Matrix2d));
        Matrix::callAllocElementH(matData, cfg.BMP_READ_DIM,cfg.BMP_READ_DIM);
        Matrix::Matrix2d* matLabel;
        hipHostMalloc(&matLabel,sizeof(Matrix::Matrix2d));
        Matrix::callAllocElementH(matLabel, cfg.OUTPUT_SIZE,1);
        Matrix::callAllocZero(matLabel);
        matLabel->elements[labelIndex] = 1.0f;
        (*label).push_back(matLabel);
        (*data).push_back(matData);
    }

    for (int i=0;i< cfg.CPU_THREADS;i++) {
        Matrix::Matrix2d* matT;
        hipHostMalloc(&matT,sizeof(Matrix::Matrix2d));
        Matrix::callAllocElementD(matT, cfg.BMP_READ_DIM,cfg.BMP_READ_DIM);
        buf.push_back(matT);
    }
    int index = 0;
    while(index < temp.size()){
        int threads = temp.size()-index < cfg.CPU_THREADS ? (int)temp.size()-index : cfg.CPU_THREADS;
        Reader::readBMPFiles(threads, paths, cfg.BMP_READ_SIZE, buffer, bufCuda, data, &buf, Reader::READ_GRAY, index, count);
        index+=cfg.CPU_THREADS;
    }

    hipHostFree(buffer);
    hipHostFree(paths);
    hipFree(bufCuda);

    logInfo("DATASET > read " + to_string(temp.size())+ " files for label : " + to_string(labelIndex));
    return temp.size();
}

void DenseMLP::loadDataSet() {
     string path0 = DenseMLP::cfg.TRAIN_DATA_PATH;
     int count = 0;
     for(int i=0; i< 10; i++){
         count += readDataset(path0 + "\\" + to_string(i), &dataset, &labelSet, cfg, i, count);
     }
}

void DenseMLP::execute() {

}

void DenseMLP::loadData() {

}

void DenseMLP::loadModel() {

}

void DenseMLP::train() {

}

void DenseMLP::unloadData() {

}
