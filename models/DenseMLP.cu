//
// Created by DanielSun on 11/22/2021.
//

#include "DenseMLP.cuh"
#include "../io/Reader.cuh"
#include "../utils/logger.cuh"
#include "../utils/Matrix.cuh"
#include "../layers/DenseLayer.cuh"
#include <random>

int readDataset(const string& path0, vector<Matrix::Matrix2d*>* data, vector<Matrix::Matrix2d*>* label,
                 DenseMLP::Config cfg, int labelIndex, int count) {
    string* paths;
    vector<Matrix::Matrix2d *> buf;
    unsigned char* buffer;
    unsigned char* bufCuda;
    vector<string> temp = Reader::getDirFiles(path0);

    hipHostMalloc((void**)&paths, sizeof(string)*temp.size());
    hipHostMalloc((void**)&buffer, sizeof(char)*cfg.CPU_THREADS*cfg.BMP_READ_SIZE);
    hipMalloc((void**)&bufCuda, sizeof(char)*cfg.CPU_THREADS*cfg.BMP_READ_SIZE);
    for(int i=1; i< temp.size(); ++i){
        paths[i-1] = temp[i];
    }

    for (int i=0;i< temp.size();i++) {

        Matrix::Matrix2d* matData;
        hipHostMalloc(&matData,sizeof(Matrix::Matrix2d));
        Matrix::callAllocElementH(matData, cfg.INPUT_SIZE_X, cfg.INPUT_SIZE_X);
        Matrix::Matrix2d* matLabel;

        hipHostMalloc(&matLabel,sizeof(Matrix::Matrix2d));
        Matrix::callAllocElementH(matLabel, cfg.OUTPUT_SIZE,1);
        Matrix::callAllocZero(matLabel);
        matLabel->elements[labelIndex] = 1.0f;

        (*label).push_back(matLabel);
        (*data).push_back(matData);
    }

    for (int i=0;i< cfg.CPU_THREADS;i++) {
        Matrix::Matrix2d* matT;
        hipHostMalloc(&matT,sizeof(Matrix::Matrix2d));
        Matrix::callAllocElementD(matT, cfg.INPUT_SIZE_X, cfg.INPUT_SIZE_X);
        buf.push_back(matT);
    }
    int index = 0;
    while(index < temp.size()){
        int threads = temp.size()-index < cfg.CPU_THREADS ? (int)temp.size()-index : cfg.CPU_THREADS;
        Reader::readBMPFiles(threads, paths, cfg.BMP_READ_SIZE, buffer, bufCuda, data, &buf, Reader::READ_GRAY, index, count);
        index+=cfg.CPU_THREADS;
    }

    hipHostFree(buffer);
    hipHostFree(paths);
    hipFree(bufCuda);

    logInfo("DATASET > read " + to_string(temp.size())+ " files for label : " + to_string(labelIndex));
    return (int)temp.size();
}

void DenseMLP::registerModel() {
     hipHostMalloc((void**)&costBuffer, sizeof(Matrix::Matrix2d));
     Matrix::callAllocElementD(costBuffer, cfg.OUTPUT_SIZE, 1);
     logInfo("===========< REGISTERING : DenseMLP >============",0x05);
     layers.push_back(new Layer(784));  //input layer
     layers.push_back(new DenseLayer(16, 784, 16, 1));
     layers.push_back(new DenseLayer(16, 16, 10, 2));
     layers.push_back(new DenseLayer(10, 16, 10, 3));
}


void DenseMLP::loadModel() {
    logInfo("===========< LOADING : DenseMLP >============",0x05);
     if(cfg.LOAD_MODEL_FROM_SAV){
         //....
         return;
     }
    for (Layer* layer : layers){
        if(layer->getType() == "DENSE"){
            auto* temp = (DenseLayer*)layer;
            Matrix::callAllocRandom(temp->weights);
            Matrix::callAllocRandom(temp->biases);
            logInfo("layer: " + layer->getType() + " random allocated");
        }
    }
}

void DenseMLP::loadDataSet() {
     string path0 = DenseMLP::cfg.TRAIN_DATA_PATH;
     int count = 0;
     for(int i=0; i< 10; i++){
         count += readDataset(path0 + "\\" + to_string(i), &dataset, &labelSet, cfg, i, count);
     }

    for (int i=0; i< cfg.TRAIN_BATCH_SIZE; i++){
        Matrix::Matrix2d* data, *label;
        hipHostMalloc((void**)&data, sizeof(Matrix::Matrix2d));
        hipHostMalloc((void**)&label, sizeof(Matrix::Matrix2d));
        Matrix::callAllocElementD(data, cfg.INPUT_SIZE_X* cfg.INPUT_SIZE_X, 1);
        dataBatch.push_back(data);
        Matrix::callAllocElementD(label, cfg.OUTPUT_SIZE,1);
        labelBatch.push_back(label);
    }
}

void DenseMLP::run() {

}

void DenseMLP::loadData() {
    random_device rd;
    default_random_engine gen = default_random_engine(rd());
    uniform_int_distribution<int> dis(0,(int)dataset.size()-1);
    for(int i=0; i<cfg.TRAIN_BATCH_SIZE; i++){
        int index = dis(gen);
        copyH2D(dataset[index], dataBatch[i]);
        copyH2D(labelSet[index], labelBatch[i]);
    }
}

void DenseMLP::train() {
    int success=0;
    pastCost=0;
    for (int trial = 0; trial < cfg.TRAIN_BATCH_SIZE; trial++) {
        layers[0]->nodes = flattern(dataBatch[trial]);

        //forward feeding
        for (int i = 1; i < layers.size(); i++) {
            layers[i]->activate(layers[i - 1]);
        }

        //calculate cost
        costBuffer = *(*(*copyD2D(layers[layers.size()-1]->nodes, costBuffer) - labelBatch[trial])^2)*0.5;
        float cost = sumC(costBuffer);
        correctOut->nodes = labelBatch[trial];
        pastCost += cost;

        //calculate correction
        int maxIndex1 = 0, maxIndex2 = 0;
        Matrix::Matrix2d* debug;
        hipHostMalloc((void**)&debug, sizeof(Matrix::Matrix2d));
        Matrix::callAllocElementH(debug, DenseMLP::cfg.OUTPUT_SIZE, 1);
        hipMemcpy(debug->elements, layers[3]->nodes->elements, sizeof(float) * cfg.OUTPUT_SIZE, hipMemcpyDeviceToHost);
        for(int i=0; i< cfg.OUTPUT_SIZE; i++) {
            maxIndex1 = *(debug->elements + i) > *(debug->elements + maxIndex1) ? i : maxIndex1;
        }
        hipMemcpy(debug->elements, correctOut->nodes->elements, sizeof(float) * cfg.OUTPUT_SIZE, hipMemcpyDeviceToHost);
        for(int i=0; i< cfg.OUTPUT_SIZE; i++){
            maxIndex2 = *(debug->elements + i) > *(debug->elements + maxIndex2) ? i: maxIndex2;
        }
        success = maxIndex1 == maxIndex2 ? success+1 : success;

        //back propagate
        for (int i = (int)layers.size()-1; i > 0; i--) {
            layers[i]->propagate(layers[i - 1],i+1 < layers.size()? layers[i+1] : correctOut);
        }
        hipHostFree(debug->elements);
        hipHostFree(debug);
    }

    //apply changes (errors)
    for (int i = (int)layers.size()-1; i > 0; i--) {
        layers[i]->learn(cfg.TRAIN_BATCH_SIZE, cfg.LEARNING_RATE);
    }
    logInfo("Batch trained with cost: " + to_string(pastCost/(float)cfg.TRAIN_BATCH_SIZE) +
    " success rate: " + to_string(success));
}