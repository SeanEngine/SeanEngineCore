//
// Created by DanielSun on 11/22/2021.
//

#include "DenseMLP.cuh"
#include "../io/Reader.cuh"
#include "../utils/logger.cuh"
#include "../utils/Matrix.cuh"
#include "../layers/DenseLayer.cuh"
#include <random>

int readDataset(const string& path0, vector<Matrix::Matrix2d*>* data, vector<Matrix::Matrix2d*>* label,
                 DenseMLP::Config cfg, int labelIndex, int count) {
    intptr_t hFile = 0;
    struct _finddata_t fileInfo{};
    string* paths;
    vector<Matrix::Matrix2d *> buf;
    unsigned char* buffer;
    unsigned char* bufCuda;
    vector<string> temp = Reader::getDirFiles(path0);

    hipHostMalloc((void**)&paths, sizeof(string)*temp.size());
    hipHostMalloc((void**)&buffer, sizeof(char)*cfg.CPU_THREADS*cfg.BMP_READ_SIZE);
    hipMalloc((void**)&bufCuda, sizeof(char)*cfg.CPU_THREADS*cfg.BMP_READ_SIZE);
    for(int i=1; i< temp.size(); ++i){
        paths[i-1] = temp[i];
    }

    for (int i=0;i< temp.size();i++) {

        Matrix::Matrix2d* matData;
        hipHostMalloc(&matData,sizeof(Matrix::Matrix2d));
        Matrix::callAllocElementH(matData, cfg.INPUT_SIZE_X, cfg.INPUT_SIZE_X);
        Matrix::Matrix2d* matLabel;

        hipHostMalloc(&matLabel,sizeof(Matrix::Matrix2d));
        Matrix::callAllocElementH(matLabel, cfg.OUTPUT_SIZE,1);
        Matrix::callAllocZero(matLabel);
        matLabel->elements[labelIndex] = 1.0f;

        (*label).push_back(matLabel);
        (*data).push_back(matData);
    }

    for (int i=0;i< cfg.CPU_THREADS;i++) {
        Matrix::Matrix2d* matT;
        hipHostMalloc(&matT,sizeof(Matrix::Matrix2d));
        Matrix::callAllocElementD(matT, cfg.INPUT_SIZE_X, cfg.INPUT_SIZE_X);
        buf.push_back(matT);
    }
    int index = 0;
    while(index < temp.size()){
        int threads = temp.size()-index < cfg.CPU_THREADS ? (int)temp.size()-index : cfg.CPU_THREADS;
        Reader::readBMPFiles(threads, paths, cfg.BMP_READ_SIZE, buffer, bufCuda, data, &buf, Reader::READ_GRAY, index, count);
        index+=cfg.CPU_THREADS;
    }

    hipHostFree(buffer);
    hipHostFree(paths);
    hipFree(bufCuda);

    logInfo("DATASET > read " + to_string(temp.size())+ " files for label : " + to_string(labelIndex));
    return (int)temp.size();
}

void DenseMLP::registerModel() {
     logInfo("===========< REGISTERING : DenseMLP >============",0x05);
     layers.push_back(new Layer(784));  //input layer
     layers.push_back(new DenseLayer(0, 16, 784, 16, 1));
     layers.push_back(new DenseLayer(0, 16, 16, 10, 2));
     layers.push_back(new DenseLayer(0, 10, 16, 10, 3));
}


void DenseMLP::loadModel() {
    logInfo("===========< LOADING : DenseMLP >============",0x05);
     if(cfg.LOAD_MODEL_FROM_SAV){
         //....
         return;
     }
    for (Layer* layer : layers){
        if(layer->getType() == "DENSE"){
            auto* temp = (DenseLayer*)layer;
            Matrix::callAllocRandom(temp->weights);
            Matrix::callAllocRandom(temp->biases);
            logInfo("layer: " + layer->getType() + " random allocated");
        }
    }
}

void DenseMLP::loadDataSet() {
     string path0 = DenseMLP::cfg.TRAIN_DATA_PATH;
     int count = 0;
     for(int i=0; i< 10; i++){
         count += readDataset(path0 + "\\" + to_string(i), &dataset, &labelSet, cfg, i, count);
     }

    for (int i=0; i< cfg.TRAIN_BATCH_SIZE; i++){
        Matrix::Matrix2d* data, *label;
        hipHostMalloc((void**)&data, sizeof(Matrix::Matrix2d));
        hipHostMalloc((void**)&label, sizeof(Matrix::Matrix2d));
        Matrix::callAllocElementD(data, cfg.INPUT_SIZE_X, cfg.INPUT_SIZE_X);
        dataBatch.push_back(data);
        Matrix::callAllocElementD(label, cfg.OUTPUT_SIZE,1);
        labelBatch.push_back(label);
    }
}

void DenseMLP::run() {

}

void DenseMLP::loadData() {
    random_device rd;
    default_random_engine gen = default_random_engine(rd());
    uniform_int_distribution<int> dis(0,(int)dataset.size());
    cout<<dataset.size()<<endl;
    for(int i=0; i<cfg.TRAIN_BATCH_SIZE; i++){
        int index = dis(gen);
        cout<<index<<" ";
        copyH2D(dataset[index], dataBatch[i]);
        copyH2D(labelSet[index], labelBatch[i]);
    }
}

void DenseMLP::train() {

}