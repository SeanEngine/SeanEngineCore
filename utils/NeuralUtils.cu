#include "hip/hip_runtime.h"
//
// Created by DanielSun on 12/8/2021.
//

#include "NeuralUtils.cuh"
#include <hip/hip_cooperative_groups.h>
#include <cstdio>
#include <iostream>
//this will trigger an exception if the condition do not met

__inline__ __device__ float warpReduce(float val) {
    for (int mask = WARP_SIZE >> 1; mask > 0; mask >>= 1) {
        val += __shfl_xor_sync(0xffffffff, val, mask);
    }
    return val;
}

__inline__ __device__ float warpCompare(float val) {
    for (int mask = WARP_SIZE >> 1; mask > 0; mask >>= 1) {
        float temp = __shfl_xor_sync(0xffffffff, val, mask);
        val = temp > val ? temp : val;
    }
    return val;
}

//this thing only process matrices below the size of 1024 elements
//since nvidia decided not to code in their driver a way to sync all blocks
__global__ void softmax1024(int n, const float* src, float* dist){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int laneID = globalID % WARP_SIZE;
    __shared__ float buffer[CUDA_SOFTMAX_BLOCK];

    float value = globalID < n ? exp(src[globalID]) : 0;
    float reduceValue;
    buffer[globalID] = value;
    __syncthreads();

    unsigned int procSize = n;

    //cross warp reductions together with warp reduction
    while(procSize/WARP_SIZE > 0){
         reduceValue = globalID < procSize ? buffer[globalID] : 0;
         __syncthreads();
         reduceValue = warpReduce(reduceValue);
         if(laneID == 0 && globalID < procSize) buffer[globalID/WARP_SIZE] = reduceValue;
         procSize = procSize%WARP_SIZE ? procSize/WARP_SIZE + 1 : procSize/WARP_SIZE;
         __syncthreads();
    }

    //the last iteration
    reduceValue = globalID < procSize ? buffer[globalID] : 0;
    __syncthreads();
    reduceValue = warpReduce(reduceValue);
    if(laneID == 0 && globalID < procSize ) buffer[globalID/WARP_SIZE] = reduceValue;
    __syncthreads();

    if(globalID<n)
    dist[globalID] = value / buffer[0];
}

// this method will divide all elements of the matrix by the largest element
// preventing issues caused by overflowing of 32-bit floats with increasing model size.
__global__ void softmaxControlled1024(int n, const float* src, float* dist){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int laneID = globalID % WARP_SIZE;
    __shared__ float buffer[CUDA_SOFTMAX_BLOCK];
    float value = globalID < n ? src[globalID] : 0;
    float reduceValue;
    buffer[globalID] = value;
    __syncthreads();

    unsigned int procSize = n;

    //run the reduction but for the max value
    while(procSize/WARP_SIZE > 0){
        reduceValue = globalID < procSize ? buffer[globalID] : 0;
        __syncthreads();
        reduceValue = warpCompare(reduceValue);
        if(laneID == 0 && globalID < procSize) buffer[globalID/WARP_SIZE] = reduceValue;
        procSize = procSize%WARP_SIZE ? procSize/WARP_SIZE + 1 : procSize/WARP_SIZE;
        __syncthreads();
    }

    //the last iteration
    reduceValue = globalID < procSize ? buffer[globalID] : 0;
    __syncthreads();
    reduceValue = warpCompare(reduceValue);
    if(laneID == 0 && globalID < procSize ) buffer[globalID/WARP_SIZE] = reduceValue;
    __syncthreads();

    float MAX_VALUE = buffer[0];
    value = globalID < n ? exp(value - MAX_VALUE) : 0;
    buffer[globalID] = value;
    __syncthreads();

    procSize = n;

    //cross warp reductions together with warp reduction
    while(procSize/WARP_SIZE > 0){
        reduceValue = globalID < procSize ? buffer[globalID] : 0;
        __syncthreads();
        reduceValue = warpReduce(reduceValue);
        if(laneID == 0 && globalID < procSize) buffer[globalID/WARP_SIZE] = reduceValue;
        procSize = procSize%WARP_SIZE ? procSize/WARP_SIZE + 1 : procSize/WARP_SIZE;
        __syncthreads();
    }

    //the last iteration
    reduceValue = globalID < procSize ? buffer[globalID] : 0;
    __syncthreads();
    reduceValue = warpReduce(reduceValue);
    if(laneID == 0 && globalID < procSize ) buffer[globalID/WARP_SIZE] = reduceValue;
    __syncthreads();

    if(globalID<n)
        dist[globalID] = value / buffer[0];
}

//store every exponents in the buffer
__global__ void softMaxPrepare(unsigned int n, float* buffer){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    if(globalID < n) buffer[globalID] = exp(buffer[globalID]);
}

//execute reduction like normally
__global__ void softMaxReduce(unsigned int n, float* buffer){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int warpID = globalID % WARP_SIZE;
    float val = globalID < n ? buffer[globalID] : 0;
    __syncthreads();
    warpReduce(val);
    if(warpID == 0) buffer[globalID/WARP_SIZE] = val;
}

//use the result on the elements
__global__ void softMaxActivate(unsigned int n, const float* buffer, float* dist){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    if(globalID < n)
    dist[globalID] = exp(dist[globalID]) / buffer[0];
}

__global__ void softMaxDerivative(Matrix::Matrix2d* mat1, Matrix::Matrix2d* correctOut, Matrix::Matrix2d* result){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    result->set( globalID, 0,mat1->get(globalID,0) - correctOut->get(globalID,0));
}

// L = - y * ln(a)
__global__ void softMaxCost(Matrix::Matrix2d* mat1, Matrix::Matrix2d* correctOut, Matrix::Matrix2d* result){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    if (mat1->get(globalID,0) <= 0) mat1->set(globalID, 0, 1e-30);
    result->set( globalID, 0,-(correctOut->get(globalID,0) * log(mat1->get(globalID,0))));
}

__global__ void sigmoidActivation(Matrix::Matrix2d *mat1) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    mat1->set(row, col, 1.0f / (1.0f + exp(-x)));
}

__device__ float sigmoidCalc(float x) {
    return 1.0f / (1.0f + exp(-x));
}

__global__ void sigmoidActivation(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, sigmoidCalc(x));
}

__global__ void sigmoidDerivative(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, sigmoidCalc(x) * (1.0f - sigmoidCalc(x)));
}


__global__ void leakyReluActivation(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, x > 0 ? x : ALPHA * x);
}

__global__ void leakyReluDerivative(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, x > 0 ? 1 : ALPHA);
}

__global__ void convPrepareFilter(Matrix::Matrix3d *filter,  Matrix::Matrix2d* filterBuffer) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int position = col % (unsigned int)pow(filter->rowcount, 2);
    filterBuffer->set(row, col, filter->get(row, position));
}

__global__ void convPrepareFeatureMap(Matrix::Matrix3d* featureMaps, Matrix::Matrix2d* featureBuffer,
                                      unsigned int filterSize, unsigned int stride){
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int depth = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int filterOffset = filterSize/2;
    unsigned int applyColCount = (featureMaps->rowcount-filterSize) / stride + 1;

    //set the copying parameters
    unsigned int convCenterRow = filterOffset + col / applyColCount * stride;
    unsigned int convCenterCol = filterOffset + col % applyColCount * stride;

    //copy feature maps to buffer
    for(unsigned int i=convCenterRow - filterOffset; i <= convCenterRow + filterOffset; i++){
        for(unsigned int j=convCenterCol - filterOffset; j <= convCenterCol + filterOffset; j++){
             featureBuffer->set(depth*filterSize*filterSize + i*filterSize+j, col, featureMaps->get(depth,i,j));
        }
    }
}

//activations
Matrix::Matrix2d *NeuralUtils::callActivationSigmoid(Matrix::Matrix2d *mat1) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    sigmoidActivation<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1);
    hipDeviceSynchronize();
    return mat1;
}

Matrix::Matrix2d *NeuralUtils::callActivationSigmoid(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    sigmoidActivation<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callDerivativeSigmoid(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    sigmoidDerivative<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callLeakyReluDerivative(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    leakyReluDerivative<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result, ALPHA);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callLeakyReluActivation(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    leakyReluActivation<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result, ALPHA);
    hipDeviceSynchronize();
    return result;
}

//buffer can be set to null if the softmax operation is applied to matrices less than 1024 elements
//call the softmax activation
Matrix::Matrix2d *NeuralUtils::callSoftMax(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float* buffer) {
    assert(mat1->rowcount * mat1->colcount == result->rowcount * result->colcount);
    unsigned int n =  mat1->rowcount * mat1->colcount;
    unsigned int gridSize = n/ (CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y) + 1;
    unsigned int blockSize = CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y;
    if(n <= 1024) {
        softmaxControlled1024<<<1, CUDA_SOFTMAX_BLOCK>>>(n, mat1->elements, result->elements);
        hipDeviceSynchronize();
        return result;
    }
    assert(buffer != nullptr);
    hipMemcpy(buffer, mat1->elements, sizeof(float) *n, hipMemcpyDeviceToDevice);
    softMaxPrepare<<<gridSize, blockSize>>>(n, buffer);
    hipDeviceSynchronize();
    unsigned int procSize = n;
    while(procSize/WARP_SIZE > 0){
        softMaxReduce<<<gridSize, blockSize>>>(procSize, buffer);
        procSize = procSize%WARP_SIZE ? procSize/WARP_SIZE + 1 : procSize/WARP_SIZE;
        hipDeviceSynchronize();
    }
    softMaxReduce<<<gridSize,blockSize>>>(procSize, buffer);
    hipDeviceSynchronize();
    softMaxActivate<<<gridSize, blockSize>>>(n, buffer, result->elements);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *
NeuralUtils::callSoftMaxDerivatives(Matrix::Matrix2d *mat1, Matrix::Matrix2d *correctOut, Matrix::Matrix2d *result) {
    assert(mat1->rowcount == correctOut->rowcount && mat1->rowcount == result->rowcount);
    assert(mat1->colcount == 1 && result->colcount == 1 && correctOut->colcount==1);
    unsigned int n =  mat1->rowcount;
    unsigned int gridSize = n/ (CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y) + 1;
    unsigned int blockSize = CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y;
    softMaxDerivative<<<gridSize, blockSize>>>(mat1, correctOut, result);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callSoftMaxCost(Matrix::Matrix2d *mat1,Matrix::Matrix2d *correctOut, Matrix::Matrix2d *result) {
    unsigned int n =  mat1->rowcount;
    unsigned int gridSize = n/ (CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y) + 1;
    unsigned int blockSize = CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y;
    softMaxCost<<<gridSize, blockSize>>>(mat1, correctOut, result);
    hipDeviceSynchronize();
    return result;
}
// filter: rowcount = colcount
// see : https://sahnimanas.github.io/post/anatomy-of-a-high-performance-convolution/
//outputDim = (n-f)/s + 1
Matrix::Matrix3d *
NeuralUtils::callConv2d(Matrix::Matrix3d *mat1, Matrix::Matrix3d *filter, Matrix::Matrix3d *result, unsigned int stride,
                        Matrix::Matrix2d* filterBuffer, Matrix::Matrix2d* featureBuffer) {
    assert(mat1->rowcount == mat1->colcount && filter->rowcount==filter->colcount);
    assert(mat1->rowcount-filter->rowcount % stride == 0);
    assert(filterBuffer->rowcount == filter->depthCount && filterBuffer->colcount == filter->colcount*filter->colcount*mat1->depthCount);
    assert(result->rowcount == (mat1->rowcount-filter->rowcount) / stride + 1);
    assert(result->colcount == result->rowcount && result->depthCount == filter->depthCount);
    assert(featureBuffer->colcount==filterBuffer->rowcount * filterBuffer->colcount * mat1->depthCount
         && featureBuffer->rowcount == filterBuffer->colcount);

    dim3 filterGridSize = dim3((filterBuffer->colcount + CUDA_BLOCK_SIZE.x-1)/CUDA_BLOCK_SIZE.x,
                               (filterBuffer->rowcount + CUDA_BLOCK_SIZE.y-1)/CUDA_BLOCK_SIZE.y);
    dim3 featureGridSize = dim3((featureBuffer->colcount + CUDA_BLOCK_SIZE.x-1)/CUDA_BLOCK_SIZE.x,
                                (featureBuffer->rowcount/(filter->rowcount * filter->colcount) + CUDA_BLOCK_SIZE.y-1)/ CUDA_BLOCK_SIZE.y);
    convPrepareFilter<<<filterGridSize, CUDA_BLOCK_SIZE>>>(filter, filterBuffer);
    convPrepareFeatureMap<<<featureGridSize, CUDA_BLOCK_SIZE>>>(mat1,featureBuffer, filter->colcount, stride);
    Matrix::Matrix2d* tmp;
    hipHostMalloc(&tmp, sizeof(Matrix::Matrix2d));
    tmp->rowcount = filter->depthCount;
    tmp->colcount = featureBuffer->colcount;
    tmp->elements = result->elements;
    hipDeviceSynchronize();
    cross(filterBuffer, featureBuffer, tmp);
    return result;
}
