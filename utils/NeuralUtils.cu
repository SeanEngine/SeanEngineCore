#include "hip/hip_runtime.h"
//
// Created by DanielSun on 12/8/2021.
//

#include "NeuralUtils.cuh"
#include <hip/hip_cooperative_groups.h>
#include <cstdio>
#include <iostream>
//this will trigger an exception if the condition do not met
#define cuAssert(condition) if(!(condition)){ asm{"trap"}; }

__inline__ __device__ float warpReduce(float val) {
    for (int mask = WARP_SIZE >> 1; mask > 0; mask >>= 1) {
        val += __shfl_xor_sync(0xffffffff, val, mask);
    }
    return val;
}

//this thing only process matrices below the size of 1024 elements
//since nvidia decided not to code in their driver a way to sync all blocks
__global__ void softmax1024(int n, const float* src, float* dist){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int laneID = globalID % WARP_SIZE;
    __shared__ float buffer[CUDA_SOFTMAX_BLOCK];

    float value = globalID < n ? exp(src[globalID]) : 0;
    float reduceValue;
    buffer[globalID] = value;
    __syncthreads();

    int procSize = n;

    //cross warp reductions together with warp reduction
    while(procSize/WARP_SIZE > 0){
         reduceValue = globalID < procSize ? buffer[globalID] : 0;
         __syncthreads();
         reduceValue = warpReduce(reduceValue);
         if(laneID == 0 && globalID < procSize) buffer[globalID/WARP_SIZE] = reduceValue;
         procSize = procSize%WARP_SIZE ? procSize/WARP_SIZE + 1 : procSize/WARP_SIZE;
         __syncthreads();
    }

    //the last iteration
    reduceValue = globalID < procSize ? buffer[globalID] : 0;
    __syncthreads();
    reduceValue = warpReduce(reduceValue);
    if(laneID == 0 && globalID < procSize ) buffer[globalID/WARP_SIZE] = reduceValue;
    __syncthreads();

    dist[globalID] = value / buffer[0];
}

//store every exponents in the buffer
__global__ void softMaxPrepare(int n, float* buffer){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    if(globalID < n) buffer[globalID] = exp(buffer[globalID]);
}

//execute reduction like normally
__global__ void softMaxReduce(int n, float* buffer){
    int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    int warpID = globalID % WARP_SIZE;
    float val = globalID < n ? buffer[globalID] : 0;
    __syncthreads();
    warpReduce(val);
    if(warpID == 0) buffer[globalID/WARP_SIZE] = val;
}

//use the result on the elements
__global__ void softMaxActivate(int n, const float* buffer, float* dist){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    if(globalID < n)
    dist[globalID] = exp(dist[globalID]) / buffer[0];
}

__global__ void sigmoidActivation(Matrix::Matrix2d *mat1) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    mat1->set(row, col, 1.0f / (1.0f + exp(-x)));
}

__device__ float sigmoidCalc(float x) {
    return 1.0f / (1.0f + exp(-x));
}

__global__ void sigmoidActivation(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, sigmoidCalc(x));
}

__global__ void sigmoidDerivative(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, sigmoidCalc(x) * (1.0f - sigmoidCalc(x)));
}


__global__ void leakyReluActivation(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, x > 0 ? x : ALPHA * x);
}

__global__ void leakyReluDerivative(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, x > 0 ? 1 : ALPHA);
}

//activations
Matrix::Matrix2d *NeuralUtils::callActivationSigmoid(Matrix::Matrix2d *mat1) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    sigmoidActivation<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1);
    hipDeviceSynchronize();
    return mat1;
}

Matrix::Matrix2d *NeuralUtils::callActivationSigmoid(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    sigmoidActivation<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callDerivativeSigmoid(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    sigmoidDerivative<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callLeakyReluDerivative(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    leakyReluDerivative<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result, ALPHA);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callLeakyReluActivation(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    leakyReluActivation<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result, ALPHA);
    hipDeviceSynchronize();
    return result;
}

//buffer can be set to null if the softmax operation is applied to matrices less than 1024 elements
//call the softmax activation
Matrix::Matrix2d *NeuralUtils::callSoftMax(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float* buffer) {
    assert(mat1->rowcount * mat1->colcount == result->rowcount * result->colcount);
    int n =  mat1->rowcount * mat1->colcount;
    unsigned int gridSize = n/ (CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y) + 1;
    unsigned int blockSize = CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y;
    if(n <= 1024) {
        softmax1024<<<1, CUDA_SOFTMAX_BLOCK>>>(n, mat1->elements, result->elements);
        hipDeviceSynchronize();
        return result;
    }
    assert(buffer != nullptr);
    hipMemcpy(buffer, mat1->elements, sizeof(float) *n, hipMemcpyDeviceToDevice);
    softMaxPrepare<<<gridSize, blockSize>>>(n, buffer);
    hipDeviceSynchronize();
    int procSize = n;
    while(procSize/WARP_SIZE > 0){
        softMaxReduce<<<gridSize, blockSize>>>(procSize, buffer);
        procSize = procSize%WARP_SIZE ? procSize/WARP_SIZE + 1 : procSize/WARP_SIZE;
        hipDeviceSynchronize();
    }
    softMaxReduce<<<gridSize,blockSize>>>(procSize, buffer);
    hipDeviceSynchronize();
    softMaxActivate<<<gridSize, blockSize>>>(n, buffer, result->elements);
    hipDeviceSynchronize();
    return result;
}
