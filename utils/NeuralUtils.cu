#include "hip/hip_runtime.h"
//
// Created by DanielSun on 12/8/2021.
//

#include "NeuralUtils.cuh"
#include <hip/hip_cooperative_groups.h>
#include <cstdio>
#include <iostream>
//this will trigger an exception if the condition do not met

__inline__ __device__ float warpReduce(float val) {
    for (int mask = WARP_SIZE >> 1; mask > 0; mask >>= 1) {
        val += __shfl_xor_sync(0xffffffff, val, mask);
    }
    return val;
}

__inline__ __device__ float warpCompare(float val) {
    for (int mask = WARP_SIZE >> 1; mask > 0; mask >>= 1) {
        float temp = __shfl_xor_sync(0xffffffff, val, mask);
        val = temp > val ? temp : val;
    }
    return val;
}

//this thing only process matrices below the size of 1024 elements
//since nvidia decided not to code in their driver a way to sync all blocks
__global__ void softmax1024(int n, const float* src, float* dist){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int laneID = globalID % WARP_SIZE;
    __shared__ float buffer[CUDA_SOFTMAX_BLOCK];

    float value = globalID < n ? exp(src[globalID]) : 0;
    float reduceValue;
    buffer[globalID] = value;
    __syncthreads();

    unsigned int procSize = n;

    //cross warp reductions together with warp reduction
    while(procSize/WARP_SIZE > 0){
         reduceValue = globalID < procSize ? buffer[globalID] : 0;
         __syncthreads();
         reduceValue = warpReduce(reduceValue);
         if(laneID == 0 && globalID < procSize) buffer[globalID/WARP_SIZE] = reduceValue;
         procSize = procSize%WARP_SIZE ? procSize/WARP_SIZE + 1 : procSize/WARP_SIZE;
         __syncthreads();
    }

    //the last iteration
    reduceValue = globalID < procSize ? buffer[globalID] : 0;
    __syncthreads();
    reduceValue = warpReduce(reduceValue);
    if(laneID == 0 && globalID < procSize ) buffer[globalID/WARP_SIZE] = reduceValue;
    __syncthreads();

    if(globalID<n)
    dist[globalID] = value / buffer[0];
}

// this method will divide all elements of the matrix by the largest element
// preventing issues caused by overflowing of 32-bit floats with increasing model size.
__global__ void softmaxControlled1024(int n, const float* src, float* dist){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int laneID = globalID % WARP_SIZE;
    __shared__ float buffer[CUDA_SOFTMAX_BLOCK];
    float value = globalID < n ? src[globalID] : 0;
    float reduceValue;
    buffer[globalID] = value;
    __syncthreads();

    unsigned int procSize = n;

    //run the reduction but for the max value
    while(procSize/WARP_SIZE > 0){
        reduceValue = globalID < procSize ? buffer[globalID] : 0;
        __syncthreads();
        reduceValue = warpCompare(reduceValue);
        if(laneID == 0 && globalID < procSize) buffer[globalID/WARP_SIZE] = reduceValue;
        procSize = procSize%WARP_SIZE ? procSize/WARP_SIZE + 1 : procSize/WARP_SIZE;
        __syncthreads();
    }

    //the last iteration
    reduceValue = globalID < procSize ? buffer[globalID] : 0;
    __syncthreads();
    reduceValue = warpCompare(reduceValue);
    if(laneID == 0 && globalID < procSize ) buffer[globalID/WARP_SIZE] = reduceValue;
    __syncthreads();

    float MAX_VALUE = buffer[0];
    value = globalID < n ? exp(value - MAX_VALUE) : 0;
    buffer[globalID] = value;
    __syncthreads();

    procSize = n;

    //cross warp reductions together with warp reduction
    while(procSize/WARP_SIZE > 0){
        reduceValue = globalID < procSize ? buffer[globalID] : 0;
        __syncthreads();
        reduceValue = warpReduce(reduceValue);
        if(laneID == 0 && globalID < procSize) buffer[globalID/WARP_SIZE] = reduceValue;
        procSize = procSize%WARP_SIZE ? procSize/WARP_SIZE + 1 : procSize/WARP_SIZE;
        __syncthreads();
    }

    //the last iteration
    reduceValue = globalID < procSize ? buffer[globalID] : 0;
    __syncthreads();
    reduceValue = warpReduce(reduceValue);
    if(laneID == 0 && globalID < procSize ) buffer[globalID/WARP_SIZE] = reduceValue;
    __syncthreads();

    if(globalID<n)
        dist[globalID] = value / buffer[0];
}

//store every exponents in the buffer
__global__ void softMaxPrepare(int n, float* buffer){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    if(globalID < n) buffer[globalID] = exp(buffer[globalID]);
}

//execute reduction like normally
__global__ void softMaxReduce(int n, float* buffer){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int warpID = globalID % WARP_SIZE;
    float val = globalID < n ? buffer[globalID] : 0;
    __syncthreads();
    warpReduce(val);
    if(warpID == 0) buffer[globalID/WARP_SIZE] = val;
}

//use the result on the elements
__global__ void softMaxActivate(int n, const float* buffer, float* dist){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    if(globalID < n)
    dist[globalID] = exp(dist[globalID]) / buffer[0];
}

__global__ void softMaxDerivative(Matrix::Matrix2d* mat1, Matrix::Matrix2d* correctOut, Matrix::Matrix2d* result){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    result->set( globalID, 0,mat1->get(globalID,0) - correctOut->get(globalID,0));
}

// L = - y * ln(a)
__global__ void softMaxCost(Matrix::Matrix2d* mat1, Matrix::Matrix2d* correctOut, Matrix::Matrix2d* result){
    unsigned int globalID = threadIdx.x + blockIdx.x * blockDim.x;
    if (mat1->get(globalID,0) <= 0) mat1->set(globalID, 0, 1e-30);
    result->set( globalID, 0,-(correctOut->get(globalID,0) * log(mat1->get(globalID,0))));
}

__global__ void sigmoidActivation(Matrix::Matrix2d *mat1) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    mat1->set(row, col, 1.0f / (1.0f + exp(-x)));
}

__device__ float sigmoidCalc(float x) {
    return 1.0f / (1.0f + exp(-x));
}

__global__ void sigmoidActivation(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, sigmoidCalc(x));
}

__global__ void sigmoidDerivative(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, sigmoidCalc(x) * (1.0f - sigmoidCalc(x)));
}


__global__ void leakyReluActivation(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, x > 0 ? x : ALPHA * x);
}

__global__ void leakyReluDerivative(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, x > 0 ? 1 : ALPHA);
}

//activations
Matrix::Matrix2d *NeuralUtils::callActivationSigmoid(Matrix::Matrix2d *mat1) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    sigmoidActivation<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1);
    hipDeviceSynchronize();
    return mat1;
}

Matrix::Matrix2d *NeuralUtils::callActivationSigmoid(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    sigmoidActivation<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callDerivativeSigmoid(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    sigmoidDerivative<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callLeakyReluDerivative(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    leakyReluDerivative<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result, ALPHA);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callLeakyReluActivation(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    leakyReluActivation<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result, ALPHA);
    hipDeviceSynchronize();
    return result;
}

//buffer can be set to null if the softmax operation is applied to matrices less than 1024 elements
//call the softmax activation
Matrix::Matrix2d *NeuralUtils::callSoftMax(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float* buffer) {
    assert(mat1->rowcount * mat1->colcount == result->rowcount * result->colcount);
    int n =  mat1->rowcount * mat1->colcount;
    unsigned int gridSize = n/ (CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y) + 1;
    unsigned int blockSize = CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y;
    if(n <= 1024) {
        softmaxControlled1024<<<1, CUDA_SOFTMAX_BLOCK>>>(n, mat1->elements, result->elements);
        hipDeviceSynchronize();
        return result;
    }
    assert(buffer != nullptr);
    hipMemcpy(buffer, mat1->elements, sizeof(float) *n, hipMemcpyDeviceToDevice);
    softMaxPrepare<<<gridSize, blockSize>>>(n, buffer);
    hipDeviceSynchronize();
    int procSize = n;
    while(procSize/WARP_SIZE > 0){
        softMaxReduce<<<gridSize, blockSize>>>(procSize, buffer);
        procSize = procSize%WARP_SIZE ? procSize/WARP_SIZE + 1 : procSize/WARP_SIZE;
        hipDeviceSynchronize();
    }
    softMaxReduce<<<gridSize,blockSize>>>(procSize, buffer);
    hipDeviceSynchronize();
    softMaxActivate<<<gridSize, blockSize>>>(n, buffer, result->elements);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *
NeuralUtils::callSoftMaxDerivatives(Matrix::Matrix2d *mat1, Matrix::Matrix2d *correctOut, Matrix::Matrix2d *result) {
    assert(mat1->rowcount == correctOut->rowcount && mat1->rowcount == result->rowcount);
    assert(mat1->colcount == 1 && result->colcount == 1 && correctOut->colcount==1);
    int n =  mat1->rowcount;
    unsigned int gridSize = n/ (CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y) + 1;
    unsigned int blockSize = CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y;
    softMaxDerivative<<<gridSize, blockSize>>>(mat1, correctOut, result);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callSoftMaxCost(Matrix::Matrix2d *mat1,Matrix::Matrix2d *correctOut, Matrix::Matrix2d *result) {
    int n =  mat1->rowcount;
    unsigned int gridSize = n/ (CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y) + 1;
    unsigned int blockSize = CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y;
    softMaxCost<<<gridSize, blockSize>>>(mat1, correctOut, result);
    hipDeviceSynchronize();
    return result;
}
