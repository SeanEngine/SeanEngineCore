#include "hip/hip_runtime.h"
//
// Created by DanielSun on 12/8/2021.
//

#include "NeuralUtils.cuh"


__global__ void sigmoidActivation(Matrix::Matrix2d *mat1) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    mat1->set(row, col, 1.0f / (1.0f + exp(-x)));
}

__device__ float sigmoidCalc(float x) {
    return 1.0f / (1.0f + exp(-x));
}

__global__ void sigmoidActivation(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, sigmoidCalc(x));
}

__global__ void sigmoidDerivative(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, sigmoidCalc(x) * (1.0f - sigmoidCalc(x)));
}


__global__ void leakyReluActivation(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, x > 0 ? x : ALPHA * x);
}

__global__ void leakyReluDerivative(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float x = mat1->get(row, col);
    result->set(row, col, x > 0 ? 1 : ALPHA);
}

//activations
Matrix::Matrix2d *NeuralUtils::callActivationSigmoid(Matrix::Matrix2d *mat1) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    sigmoidActivation<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1);
    hipDeviceSynchronize();
    return mat1;
}

Matrix::Matrix2d *NeuralUtils::callActivationSigmoid(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    sigmoidActivation<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callDerivativeSigmoid(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    sigmoidDerivative<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callLeakyReluDerivative(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    leakyReluDerivative<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result, ALPHA);
    hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *NeuralUtils::callLeakyReluActivation(Matrix::Matrix2d *mat1, Matrix::Matrix2d *result, float ALPHA) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    leakyReluActivation<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result, ALPHA);
    hipDeviceSynchronize();
    return result;
}