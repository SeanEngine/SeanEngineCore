#include "hip/hip_runtime.h"
//
// Created by DanielSun on 9/23/2021.
//

#include "Matrix.cuh"
#include <string>
#include <iostream>
#include <cassert>

using namespace std;

//device methods
__device__ float Matrix::Matrix2d::get(int row, int col) const   {
    return (row < rowcount && col < colcount) ? this->elements[row * this->colcount + col] : 0.0f;
}

__device__ void Matrix::Matrix2d::set(int row, int col, float value) const   {
    if(row < rowcount && col < colcount)
    this->elements[row * this->colcount + col] = value;
}

__device__ float Matrix::fasterSqrt(float in) {
    float half = 0.5f*in;
    int i = *(int*)&in;
    i = 0x5f375a86-(i>>1);
    in = *(float*)&i;
    in = in*(1.5f-half*in*in);
    return in;
}

//random number fill (0-1)
__global__ void allocRandom(long seed, Matrix::Matrix2d *mat1){
    hiprandStateXORWOW_t state;
    int row = static_cast<int>(threadIdx.y + blockIdx.y * blockDim.y);
    int col =  static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x);
    hiprand_init((row+1)*(col+1)*seed,0,0,&state);
    mat1->set(row, col, static_cast<float>((hiprand_uniform(&state) * 2.0F) - 1.0F));
}

//zero fill
__global__ void allocZero(Matrix::Matrix2d *mat1) {
    int row =  static_cast<int>(threadIdx.y + blockIdx.y * blockDim.y);
    int col =  static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x);
    mat1->set(row, col, 0.0F);
}

__global__ void crossP(Matrix::Matrix2d* mat1, Matrix::Matrix2d* mat2, Matrix::Matrix2d* result){
    float currentValue = 0.0;
    int row =  static_cast<int>(threadIdx.y + blockIdx.y * blockDim.y);
    int col =  static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x);

    for (int i = 0; i < mat1->colcount; ++i) {
        currentValue += mat1->get(row, i) *
                        mat2->get(i, col);
    }

    result->set(row, col, currentValue);
}

//constants : TILE_SIZE = blockSize.x = blockSize.y,
__global__ void crossTiling(Matrix::Matrix2d* mat1, Matrix::Matrix2d* mat2, Matrix::Matrix2d* result){

    __shared__ float mat1_tile[TILE_SIZE][TILE_SIZE];
    __shared__ float mat2_tile[TILE_SIZE][TILE_SIZE];
    float resultOutput = 0;   // result of C in register

    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    //#pragma unroll
    for (int tileId = 0; tileId < (mat1->colcount + TILE_SIZE-1)/TILE_SIZE; tileId++ ){

        //load shared memory
        mat1_tile[threadIdx.y][threadIdx.x] = mat1->get(row, threadIdx.x + tileId * TILE_SIZE);
        mat2_tile[threadIdx.y][threadIdx.x] = mat2->get(threadIdx.y + tileId * TILE_SIZE, col);
        __syncthreads();

        #pragma unroll
        for(int mulIndex = 0; mulIndex < TILE_SIZE; mulIndex++){
            resultOutput += mat1_tile[threadIdx.y][mulIndex] * mat2_tile[mulIndex][threadIdx.x];
        }
        __syncthreads();
    }

    result->set(row, col, resultOutput);
}

/*
 * This method requires the distribution of grid and blocks to be:
 *     dim3 blockSize(TILE_SIZE, VECTOR_SIZE);
 *     dim3 grid(mat2->colcount / (TILE_SIZE * VECTOR_SIZE), mat1->rowcount / TILE_SIZE);
 */

__global__ void crossCompOpt(Matrix::Matrix2d* mat1, Matrix::Matrix2d* mat2, Matrix::Matrix2d* result){

    __shared__ float mat1_tile[TILE_SIZE][TILE_SIZE];
    float mat2Value = 0.0f;
    float resultBuffer[TILE_SIZE] = {0};

    int resultCol = VECTOR_SIZE*TILE_SIZE*blockIdx.x + threadIdx.y * TILE_SIZE + threadIdx.x;

    for (int tileId = 0; tileId < (mat1->colcount + TILE_SIZE-1)/TILE_SIZE; tileId++ ){
        //allocate elements
        for (int i = 0; i < TILE_SIZE / VECTOR_SIZE; i++) {
            mat1_tile[threadIdx.x][i*VECTOR_SIZE + threadIdx.y]= mat1->
                    get(blockIdx.y * TILE_SIZE + i*VECTOR_SIZE + threadIdx.y,tileId * TILE_SIZE + threadIdx.x);
        }
        __syncthreads();

        for (int row = 0; row < TILE_SIZE; row++){
            //pick a value of mat2 and put it into the registers
            mat2Value = mat2->get(tileId * TILE_SIZE + row,resultCol);
            for (int bufId = 0; bufId < TILE_SIZE; bufId++){
                resultBuffer[bufId] += mat1_tile[row][bufId] * mat2Value;
            }
        }
        __syncthreads();
    }
    int resultRow0 = blockIdx.y * TILE_SIZE;
    for (int bufId = 0; bufId <TILE_SIZE; bufId++){
        result->set(resultRow0 + bufId, resultCol, resultBuffer[bufId]);
    }
}

__global__ void prefetchingCrossP(Matrix::Matrix2d* mat1, Matrix::Matrix2d* mat2, Matrix::Matrix2d* result){

}

//memory Control:
void Matrix::callAllocElementH(Matrix::Matrix2d *mat1, int row, int col) {
    mat1->rowcount = row;
    mat1->colcount = col;
    (void)hipHostMalloc(reinterpret_cast<void**>(&mat1->elements),row*col*sizeof(float));
}

void Matrix::callAllocElementD(Matrix::Matrix2d *mat1, int row, int col) {
    mat1->rowcount = row;
    mat1->colcount = col;
    hipMalloc(reinterpret_cast<void**>(&mat1->elements),row*col*sizeof(float));
}


void Matrix::callAllocRandom(Matrix::Matrix2d *mat1) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x-1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y-1) / CUDA_BLOCK_SIZE.y);
    allocRandom<<<gridSize, CUDA_BLOCK_SIZE>>>(time(nullptr),mat1);
    (void) hipDeviceSynchronize();
}

void Matrix::callAllocZero(Matrix::Matrix2d *mat1) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    allocZero<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1);
    (void) hipDeviceSynchronize();
}

Matrix::Matrix2d *Matrix::callCrossPOlD(Matrix::Matrix2d *mat1, Matrix::Matrix2d *mat2, Matrix::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    (void) crossP<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, mat2, result);
    (void) hipDeviceSynchronize();
    return result;
}

//method callings
Matrix::Matrix2d* Matrix::callCrossP(Matrix2d* mat1, Matrix2d* mat2, Matrix2d* result) {
    assert(CUDA_BLOCK_SIZE.x == CUDA_BLOCK_SIZE.y && TILE_SIZE == CUDA_BLOCK_SIZE.x);
    assert(mat1->colcount == mat2->rowcount && mat1->rowcount == result->rowcount && mat2->colcount == result->colcount);
    dim3 gridSize = dim3((mat2->colcount + CUDA_BLOCK_SIZE.x-1) / (CUDA_BLOCK_SIZE.x),
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y-1) / CUDA_BLOCK_SIZE.y);
    (void) crossTiling<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, mat2, result);
    (void) hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *Matrix::callCrossCompOpt(Matrix::Matrix2d *mat1, Matrix::Matrix2d *mat2, Matrix::Matrix2d *result) {
    assert(mat1->colcount == mat2->rowcount && mat1->rowcount == result->rowcount && mat2->colcount == result->colcount);
    dim3 blockSize = dim3(TILE_SIZE, VECTOR_SIZE);
    dim3 grid = dim3((mat2->colcount + (TILE_SIZE * VECTOR_SIZE)-1) /
            (TILE_SIZE * VECTOR_SIZE), (mat1->rowcount + TILE_SIZE -1) / TILE_SIZE);
    (void) crossCompOpt<<<grid, blockSize>>>(mat1, mat2, result);
    (void) hipDeviceSynchronize();
    return result;
}

Matrix::Matrix2d *Matrix::callConstantP(Matrix::Matrix2d *mat1, float con) {
    return nullptr;
}

Matrix::Matrix2d *Matrix::callAddition(Matrix::Matrix2d *mat1, Matrix::Matrix2d *mat2) {
    return nullptr;
}

Matrix::Matrix2d *Matrix::callAddition(Matrix::Matrix2d *mat1, float con) {
    return nullptr;
}

Matrix::Matrix2d *Matrix::callSubtraction(Matrix::Matrix2d *mat1, Matrix::Matrix2d *mat2) {
    return nullptr;
}

Matrix::Matrix2d *Matrix::callSubtraction(Matrix::Matrix2d *mat1, float con) {
    return nullptr;
}

Matrix::Matrix2d *Matrix::callPower(Matrix::Matrix2d *mat1, Matrix::Matrix2d *mat2) {
    return nullptr;
}

Matrix::Matrix2d *Matrix::callPower(Matrix::Matrix2d *mat1, float con) {
    return nullptr;
}

void Matrix::inspect(Matrix2d *mat1) {
    Matrix2d* debug;
    hipHostMalloc(reinterpret_cast<void**>(&debug), mat1->colcount * mat1->rowcount * sizeof(float));
    callAllocElementH(debug, mat1->rowcount, mat1->colcount);
    hipMemcpy(debug->elements,mat1->elements,sizeof(float)*debug->colcount*debug->rowcount,hipMemcpyDeviceToHost);
    for (int i = 0; i< debug -> rowcount; i++) {
        for (int j = 0; j < debug->colcount; j++) {
            std::cout<<*(debug->elements + i*debug->colcount + j)<<" ";
        }
        std::cout<<std::endl;
    }
    hipFree(debug->elements);
    hipFree(debug);
}


//operators
Matrix::Matrix2d *Matrix::Matrix2d::operator+(Matrix::Matrix2d *mat2) {
    return callAddition(this, mat2);
}

Matrix::Matrix2d *Matrix::Matrix2d::operator+(float con) {
    return callAddition(this, con);
}

Matrix::Matrix2d *Matrix::Matrix2d::operator-(Matrix::Matrix2d *mat2) {
    return callSubtraction(this, mat2);
}

Matrix::Matrix2d *Matrix::Matrix2d::operator-(float con) {
    return callSubtraction(this,con);
}

Matrix::Matrix2d *Matrix::Matrix2d::operator^(Matrix::Matrix2d *mat2) {
    return callPower(this, mat2);
}

Matrix::Matrix2d *Matrix::Matrix2d::operator^(float con) {
    return callPower(this, con);
}

Matrix::Matrix2d *Matrix::Matrix2d::operator*(float con) {
    return callConstantP(this, con);
}


