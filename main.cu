#include <cstdio>
#include "utils/Matrix.cuh"
#include <windows.h>
#include <string>
#include <iostream>
#include "io/Reader.cuh"
#include "hipblas.h"
#include "execution/ThreadControls.h"
#include "models/DenseMLP.cuh"
#include "layers/DenseLayer.cuh"
#include <opencv2/opencv.hpp>
#include <opencv2/imgcodecs.hpp>

#pragma comment(lib, "cublas.lib")

using namespace std;
using namespace nvcuda;

int main(int argc, char **argv) {

    auto *model = new DenseMLP();
    model->registerModel();
    model->loadModel();
    model->loadDataSet();
    model->loadData();
    //Matrix::inspect(((DenseLayer*)(model->layers[1]))->weights);

    for(int i=0; i<1e4; i++) {
        model->loadData();
        model->train();
    }

    int success = 0;
    for(int trial=0; trial<60000; trial++){
        Matrix::Matrix2d* data = model->dataset[trial];
        Matrix::Matrix2d* label = model->labelSet[trial];
        model->layers[0]->nodes = flattern(data);
        model->run();

        int maxIndex1 = 0, maxIndex2 = 0;
        Matrix::Matrix2d* debug;
        hipHostMalloc((void**)&debug, sizeof(Matrix::Matrix2d));
        Matrix::callAllocElementH(debug, 10, 1);
        hipMemcpy(debug->elements, model->layers[3]->nodes->elements, sizeof(float) * 10, hipMemcpyDeviceToHost);
        for(int i=0; i< 10; i++) {
            maxIndex1 = debug->elements[i] > debug->elements[maxIndex1] ? i : maxIndex1;
        }
        hipMemcpy(debug->elements, label->elements, sizeof(float) * 10, hipMemcpyDeviceToHost);
        for(int i=0; i< 10; i++){
            maxIndex2 =  debug->elements[i] > debug->elements[maxIndex2] ? i : maxIndex2;
        }
        success = maxIndex1 == maxIndex2 ? success+1 : success;
    }

    cout<<success<<endl;

    //Matrix::inspect(((DenseLayer*)(model->layers[3]))->errors);
    //Matrix::inspect(((DenseLayer*)(model->layers[3]))->nodes);
    //Matrix::inspect(model->dataBatch[model->dataBatch.size()-1]);
    //Matrix::inspect(model->labelBatch[model->labelBatch.size()-1]);
    //Matrix::inspect(model->layers[3]->nodes);

}
