#include <cstdio>
#include "utils/Matrix.cuh"
#include <windows.h>
#include <string>
#include <iostream>
#include "io/Reader.cuh"
#include "hipblas.h"
#include "execution/ThreadControls.h"
#include "models/DenseMLP.cuh"
#include "layers/DenseLayer.cuh"
#include "models/VGG16.cuh"
#include "layers/ImageContainer.cuh"
#include "layers/ConvLayer.cuh"
#include "layers/SoftmaxLayer.cuh"
#include "layers/MaxPoolingLayer.cuh"
#include <opencv2/opencv.hpp>
#include <opencv2/imgcodecs.hpp>

#pragma comment(lib, "cublas.lib")

using namespace std;
using namespace nvcuda;

void run(){

    auto* model = new VGG16();
    model->registerModel();
    model->loadModel();
    model->loadDataSet();
    for(int i=0; i<100; i++){
        logInfo("training batch ID: " + to_string(i),0x01);
        model->loadData();
        model->train();
        Matrix::inspect(((ConvLayer*)model->layers[14])->output);
    }
}

void GEMMBench(){
    Matrix::Matrix2d* A, *B, *C;
    A = Matrix::callAllocElementD(64,27);
    B = Matrix::callAllocElementD(27,50176);
    C = Matrix::callAllocElementD(64,50176);

    Matrix::callAllocRandom(A);
    Matrix::callAllocRandom(B);
    Matrix::callAllocConst(C,1);

    LARGE_INTEGER beg;
    LARGE_INTEGER end;
    LARGE_INTEGER frq;

    QueryPerformanceFrequency(&frq);
    QueryPerformanceCounter(&beg);
    crossA(A,B,C);
    cout<<endl;
    QueryPerformanceCounter(&end);
    cout<<end.QuadPart - beg.QuadPart<<endl;

    QueryPerformanceCounter(&beg);
    Matrix::callCrossPrefetching(A,B,C);
    cout<<endl;
    QueryPerformanceCounter(&end);
    cout<<end.QuadPart - beg.QuadPart<<endl;
}

int main(int argc, char **argv) {
    run();
}
