#include <cstdio>
#include "utils/Matrix.cuh"
#include <windows.h>
#include <string>
#include <iostream>
#include "hipblas.h"
#pragma comment(lib, "cublas.lib")

using namespace std;

int main(int argc, char** argv) {
     Matrix::Matrix2d* A, *B, *C;
     int TEST_SIZE=50;
     hipHostMalloc((void**)&A, sizeof(Matrix::Matrix2d));
     hipHostMalloc((void**)&B, sizeof(Matrix::Matrix2d));
     hipHostMalloc((void**)&C, sizeof(Matrix::Matrix2d));

     Matrix::callAllocElementD(A,TEST_SIZE,TEST_SIZE);
     Matrix::callAllocElementD(B, TEST_SIZE, TEST_SIZE);
     Matrix::callAllocElementD(C, TEST_SIZE, TEST_SIZE);

     Matrix::callAllocZero(A);
     float* buf;
     hipHostMalloc((void**)(&buf), sizeof(float));
     buf[0] = 0;
     sum(A, buf);
     //Matrix::inspect(A);
     cout<<*buf<<endl;
}
