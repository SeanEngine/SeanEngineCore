#include <cstdio>
#include "utils/Matrix.cuh"
#include <windows.h>
#include <string>
#include <iostream>
#include "io/Reader.cuh"
#include "hipblas.h"
#include "execution/ThreadControls.h"
#include "models/DenseMLP.cuh"

#pragma comment(lib, "cublas.lib")

using namespace std;


int main(int argc, char **argv) {

    auto *model = new DenseMLP();
    model->registerModel();
    model->loadModel();
    model->loadDataSet();
    Matrix::inspect(model->dataset[20000]);
}
