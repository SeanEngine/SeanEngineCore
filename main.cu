#include <cstdio>
#include "utils/Matrix.cuh"
#include <windows.h>
#include <string>
#include <iostream>
#include "io/Reader.cuh"
#include "hipblas.h"
#include "execution/ThreadControls.h"
#include "models/DenseMLP.cuh"
#include "layers/DenseLayer.cuh"
#include "models/VGG16.cuh"
#include "layers/ImageContainer.cuh"
#include "layers/ConvLayer.cuh"
#include "layers/SoftmaxLayer.cuh"
#include "layers/MaxPoolingLayer.cuh"
#include <opencv2/opencv.hpp>
#include <opencv2/imgcodecs.hpp>

#pragma comment(lib, "cublas.lib")

using namespace std;
using namespace nvcuda;

void run(){

    auto* model = new VGG16();
    model->registerModel();
    model->loadModel();
    model->loadDataSet();
    for(int i=0; i<1e6; i++){
        logInfo("training batch ID: " + to_string(i),0x01);
        model->loadData();
        model->train();
    }
}

int main(int argc, char **argv) {
    run();
}
